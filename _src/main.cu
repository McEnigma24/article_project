#include "hip/hip_runtime.h"
#include "__preprocessor__.h"
#include "__time_stamp__.h"

#include "hip/hip_runtime.h"
#include ""

#include "movie.h"

#include "CTRL_Scene.h"
#include "CTRL_Setuper.h"
#include "RT_Renderer.h"

#define CCE(x)                                                                                                         \
    {                                                                                                                  \
        hipError_t err = x;                                                                                           \
        if (err != hipSuccess)                                                                                        \
        {                                                                                                              \
            const string error = "CUDA ERROR - " + std::to_string(__LINE__) + " : " + __FILE__ + "\n";                 \
            cout << error;                                                                                             \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    }

__global__ void test(int* a, int* b, int* result, int ARRAY_SIZE)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (!(i < ARRAY_SIZE))
        return;

    result[i] = a[i] + b[i];
}

void OpenMP_GPU_test()
{
    int size = 1000000;
    int* a = new int[size];
    int* b = new int[size];
    int* result = new int[size];

    for (int i = 0; i < size; i++)
    {
        a[i] = i;
        b[i] = size - i;
    }

    time_stamp_reset();

    for (int i = 0; i < size; i++)
    {
        result[i] = a[i] + b[i];
    }
    time_stamp("Iterative");

#pragma omp parallel for schedule(static)
    for (int i = 0; i < size; i++)
    {
        result[i] = a[i] + b[i];
    }
    time_stamp("Parallel");

    int byte_size = size * sizeof(int);
    int* dev_a{};
    int* dev_b{};
    int* dev_result{};
    CCE(hipSetDevice(0));
    CCE(hipMalloc((void**)&dev_a, byte_size));
    CCE(hipMalloc((void**)&dev_b, byte_size));
    CCE(hipMalloc((void**)&dev_result, byte_size));

    CCE(hipMemcpy(dev_a, a, byte_size, hipMemcpyHostToDevice));
    CCE(hipMemcpy(dev_b, b, byte_size, hipMemcpyHostToDevice));

    int BLOCK_SIZE = 64;
    int NUMBER_OF_BLOCKS = size / BLOCK_SIZE + 1;

    time_stamp_reset() test<<<NUMBER_OF_BLOCKS, BLOCK_SIZE>>>(dev_a, dev_b, dev_result, size);
    CCE(hipDeviceSynchronize());
    time_stamp("GPU");

    CCE(hipMemcpy(result, dev_result, byte_size, hipMemcpyDeviceToHost));
    CCE(hipFree(dev_a));
    CCE(hipFree(dev_b));
    CCE(hipFree(dev_result));
}

// unit z_plane = u(100);

// scene.add_light
// (
//     d3(u(G::WIDTH / 2), u(G::HEIGHT / 2), z_plane)
//     ,RGB(255, 255, 255)
// );

// scene.add_sphere
// (
//     d3(u(G::WIDTH / 2), u(G::HEIGHT / 2), z_plane), u(100),		// DEAD CENTER
//     0.0f, 0.0f, Surface_type::diffuse,
//     RGB(0, 255, 0)
// );

// #define def_WIDTH (1024)
// #define def_HEIGHT (768)

#define def_WIDTH (1000)
#define def_HEIGHT (1000)
#define def_convert_2d_to_1d(x, y) (y * def_WIDTH + x)

#define FRAMES (1)

class Movie_Maker_Controller
{
    vector<uint8_t> frame_buffer;
    vector<vector<RGB>> saved_frames;

    void fill_frame_buffer(const vector<RGB>& render_output)
    {
        for (unsigned int y = 0; y < def_HEIGHT; y++)
            for (unsigned int x = 0; x < def_WIDTH; x++)
            {
                frame_buffer[4 * def_WIDTH * y + 4 * x + 2] = render_output[def_convert_2d_to_1d(x, y)].get_r();
                frame_buffer[4 * def_WIDTH * y + 4 * x + 1] = render_output[def_convert_2d_to_1d(x, y)].get_g();
                frame_buffer[4 * def_WIDTH * y + 4 * x + 0] = render_output[def_convert_2d_to_1d(x, y)].get_b();
            }
    }

public:
    Movie_Maker_Controller() : frame_buffer(4 * def_WIDTH * def_HEIGHT) {}

    void add_new_frame(const vector<RGB>& frame) { saved_frames.push_back(frame); }

    void combine_to_movie(const string& name, int frame_rate = 1)
    {
        MovieWriter movie_writer(name, def_WIDTH, def_HEIGHT, frame_rate);

        // int how_many_added_frames{};
        // for (int i = 0; i < saved_frames.size(); i++)
        // {
        //     memset(frame_buffer.data(), 0, 4 * def_WIDTH * def_HEIGHT);
        //     fill_frame_buffer(saved_frames[i]);

        //     if (i == 0 || i == saved_frames.size() - 1)
        //         how_many_added_frames = 30;
        //     else
        //         how_many_added_frames = 5;

        //     for (int ii; ii < FRAMES * how_many_added_frames; ii++)
        //         movie_writer.addFrame(&frame_buffer[0]);
        // }
    }

    void delete_all_collected_frames() { saved_frames.clear(); }
};

void fill_frame_buffer(const vector<RGB>& render_output, vector<u8>& frame_buffer)
{
    for (unsigned int y = 0; y < def_HEIGHT; y++)
        for (unsigned int x = 0; x < def_WIDTH; x++)
        {
            frame_buffer[4 * def_WIDTH * y + 4 * x + 2] = render_output[def_convert_2d_to_1d(x, y)].get_r();
            frame_buffer[4 * def_WIDTH * y + 4 * x + 1] = render_output[def_convert_2d_to_1d(x, y)].get_g();
            frame_buffer[4 * def_WIDTH * y + 4 * x + 0] = render_output[def_convert_2d_to_1d(x, y)].get_b();
        }
}

void crash_function()
{
    int* a{};
    *a = 0;
}

int main(int argc, char* argv[])
{
    srand(time(NULL));
    time_stamp("It just works");
    Movie_Maker_Controller maker;

    crash_function();

    maker.combine_to_movie("my_maker.mp4");

    return 0;

    Setuper::setup_Global_Variables___and___Clear_Stats();
    Renderer render(def_WIDTH, def_HEIGHT);

    if (false)
    {
        MovieWriter movie_writer("random_pixels.mp4", def_WIDTH, def_HEIGHT, 2);
        vector<uint8_t> frame_buffer(4 * def_WIDTH * def_HEIGHT);
        memset(frame_buffer.data(), 0, 4 * def_WIDTH * def_HEIGHT);

        Setuper::setup_Global_Variables___and___Clear_Stats();
        Renderer render(def_WIDTH, def_HEIGHT);

        {
            Scene scene;
            Setuper::setup_scene_0(&scene, "first");
            G::Render::current_scene = &scene;
            render.RENDER();
            line("rendering");

            fill_frame_buffer(render.get_my_pixel_vec(), frame_buffer);
            for (int ii; ii < FRAMES * 30; ii++)
                movie_writer.addFrame(&frame_buffer[0]);
        }

        {
            Scene scene;
            Setuper::setup_scene_1(&scene, "first");
            G::Render::current_scene = &scene;
            render.RENDER();
            line("rendering");

            fill_frame_buffer(render.get_my_pixel_vec(), frame_buffer);
            for (int ii; ii < FRAMES * 5; ii++)
                movie_writer.addFrame(&frame_buffer[0]);
        }

        {
            Scene scene;
            Setuper::setup_scene_2(&scene, "first");
            G::Render::current_scene = &scene;
            render.RENDER();
            line("rendering");

            fill_frame_buffer(render.get_my_pixel_vec(), frame_buffer);
            for (int ii; ii < FRAMES * 5; ii++)
                movie_writer.addFrame(&frame_buffer[0]);
        }

        movie_writer.addFrame(&frame_buffer[0]);
        {
            Scene scene;
            Setuper::setup_scene_6(&scene, "first");
            G::Render::current_scene = &scene;
            render.RENDER();
            line("rendering");

            fill_frame_buffer(render.get_my_pixel_vec(), frame_buffer);
            for (int ii; ii < FRAMES * 5; ii++)
                movie_writer.addFrame(&frame_buffer[0]);
        }

        {
            Scene scene;
            Setuper::setup_scene_3(&scene, "first");
            G::Render::current_scene = &scene;
            render.RENDER();
            line("rendering");

            fill_frame_buffer(render.get_my_pixel_vec(), frame_buffer);
            for (int ii; ii < FRAMES * 30; ii++)
                movie_writer.addFrame(&frame_buffer[0]);
        }
    }

    if (false)
    {
        {
            Scene scene;
            Setuper::setup_scene_0(&scene, "first");
            G::Render::current_scene = &scene;

            render.RENDER();
            maker.add_new_frame(render.get_my_pixel_vec());
            line("here");
        }

        {
            Scene scene;
            Setuper::setup_scene_1(&scene, "first");
            G::Render::current_scene = &scene;

            render.RENDER();
            maker.add_new_frame(render.get_my_pixel_vec());
        }

        {
            Scene scene;
            Setuper::setup_scene_2(&scene, "first");
            G::Render::current_scene = &scene;

            render.RENDER();
            maker.add_new_frame(render.get_my_pixel_vec());
        }

        {
            Scene scene;
            Setuper::setup_scene_3(&scene, "first");
            G::Render::current_scene = &scene;

            render.RENDER();
            maker.add_new_frame(render.get_my_pixel_vec());
        }

        line("here 1");
        maker.combine_to_movie("my_maker.mp4");
        line("here 2");
        maker.delete_all_collected_frames();
        line("here 3");
    }

    return 0;
}
